#include "hip/hip_runtime.h"
/* saving 2D arrays to disk */
void saveArrToDrive(double* f, char* outFileName)
{
    const uint16_t sampleSkip = ceil(double(N)/256.0f);
    std::ofstream ofs(outFileName);
    for(int i=0;i<N;i+=sampleSkip){
        for(int j=0;j<N;j+=sampleSkip){
            if(j>0)
                ofs << "\t";
            ofs << f[i+N*j]; 
        }
        ofs << "\n";
    }
    ofs.close();
}
/* saving interpolation to disk */
void saveIntVecToDrive(double* f, char* outFileName)
{
    const uint16_t sampleSkip = 1;
    std::ofstream ofs(outFileName);
    for(int i=0;i<M;i+=sampleSkip){
        if(i>0)
            ofs << "\t";
        ofs << f[i]; 
    }
    ofs << "\n";
    ofs.close();
}
/* saving vector to disk */
void saveVecToDrive(double* f, char* outFileName)
{
    const uint16_t sampleSkip = ceil(double(N)/256.0f);
    std::ofstream ofs(outFileName);
    for(int i=0;i<N;i+=sampleSkip){
        if(i>0)
            ofs << "\t";
        ofs << f[i]; 
    }
    ofs << "\n";
    ofs.close();
}
/* saving n-vector to disk */
void saveNVecToDrive(double* f, char* outFileName, int n)
{
    std::ofstream ofs(outFileName);
    for(int i=0;i<n;i+=1){
        if(i>0)
            ofs << "\t";
        ofs << f[i]; 
    }
    ofs << "\n";
    ofs.close();
}
/* kernel function */
#define fLJ(r,sigma) (4*U*(12*pow(sigma,12)/pow(r,13)-6*pow(sigma,6)/pow(r,7)))
#define g(r,d,sigmaC) (8.41e4*exp(-pow(r-d,2)/(2*pow(sigmaC,2))))
void genConvKernel(){
    // compute effective potential 
    double kernelL = (double(kernelN)-1)*IZ/subDiv;
    double kernelDZ = kernelL/double(kernelN-1);
    double subRes = 10000;
    double fineRes = subRes*(double(kernelN+1)/2);
    double force;
    double fineR;
    double gpdf;
    double kernelFine[int(fineRes)];
    double fineDR = kernelDZ/subRes; // only take positive values
    double sigma = 5.6e-6;
    double sigmaC = 0.5e-6;
    double eqDist = 6.58546720106423709125472581993321341542468871921300888061523437500000000000000000e-06;
    // use central interval positions
    double sum = 0;
    kernelFine[0] = 0; // avoid divergence of force term at zero
    for(int i=1;i<fineRes;i++){
        fineR = double(i*fineDR);
        force = fLJ(fineR,sigma);
        gpdf = g(fineR,eqDist,sigmaC);
        if(fineR<1e-8) // make up for numerical error near divergence
            gpdf = 0.0;
        kernelFine[i] = sum; // compute integral
        sum = sum + fineDR*force*gpdf;
    }
    
    // integration constant
    for(int i=0;i<fineRes;i++)
        kernelFine[i] = kernelFine[int(fineRes)-1]-kernelFine[i];
    // sampling of kernel
    intKernel[(kernelN+1)/2] = 0;
    double kernelZ;
    for(int i=(kernelN+1)/2;i<kernelN;i++){
        kernelZ = double(i*kernelDZ)-kernelL/2;
        intKernel[i] = kernelZ*kernelFine[int((i+1-double(kernelN+1)/2)*subRes)];
        intKernel[kernelN-1-i] = -intKernel[i];    
    }
    printf("kernel length = %.32e m\n",kernelL);
}
/* check cuda device */
inline
hipError_t checkCuda(hipError_t result)
{
    #if defined(DEBUG) || defined(_DEBUG)
        if (result != hipSuccess) {
            fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
            assert(result == hipSuccess);
        }
    #endif
    return result;
}
/* initial values for phi */
void initPhi(double *f, double *R)
{
    double edgeZ = wingL+2;
    double edgeR = wingL;
    for(int i=0;i<N;i++)
        for(int j=0;j<N;j++){
            f[i+N*j] = exp(-pow(R[j]-(Rmu),2)/(2.0*pow(Rsigma,2)));
            if((i<edgeZ)|(i>(N-1-edgeZ)))
                f[i+N*j] = 0.0;
            if((j<edgeR)|(j>(N-1-edgeR)))
                f[i+N*j] = 0.0;
        }
    // normalization
    /*
    integral phi dz drho = intgral psi dz = L N <psi> = L N PSI
    sum phi IZ = N <psi> = N PSI
    */
    double phiSum = 0.0;
    for(int i=0;i<N;i++)
        for(int j=0;j<N;j++)
            phiSum += f[i+N*j];
    for(int i=0;i<N;i++)
        for(int j=0;j<N;j++)
            f[i+N*j] = f[i+N*j]/phiSum*PSI*(N-2*edgeZ);
}
/* taking arguments */
void readParameters(int argc, char *argv[]){
    int argIdx = 1;
    // U
    if(argc>argIdx)
        U = std::stod(argv[argIdx]);
    argIdx++;
    // PSI
    if(argc>argIdx)
        PSI = std::stod(argv[argIdx]);
    argIdx++;
    // IT
    if(argc>argIdx)
        IT = std::stod(argv[argIdx]);
    argIdx++;
    // T
    if(argc>argIdx)
        T = std::stod(argv[argIdx]);
    argIdx++;
    // NO
    if(argc>argIdx)
        NO = std::stod(argv[argIdx]);
    argIdx++;
    // gamma
    if(argc>argIdx)
        h_gamma = std::stod(argv[argIdx]);
    argIdx++;
    // delta
    if(argc>argIdx)
        h_delta = std::stod(argv[argIdx]);
    argIdx++;
    // kappa
    if(argc>argIdx)
        h_kappa = std::stod(argv[argIdx]);
    argIdx++;
    // re-evalutate parameters
    NT = ceil(T/IT);
}
/* running simulation */
void runSim(){
    // allocate space for output filename
    char outFileName[19];
    // constants to device memory
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(c_IZ), &IZ, sizeof(double), 0, hipMemcpyHostToDevice) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(c_IT), &IT, sizeof(double), 0, hipMemcpyHostToDevice) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(c_PSI), &PSI, sizeof(double), 0, hipMemcpyHostToDevice) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(c_beta), &h_beta, sizeof(double), 0, hipMemcpyHostToDevice) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(c_alpha), &h_alpha, sizeof(double), 0, hipMemcpyHostToDevice) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(c_gamma), &h_gamma, sizeof(double), 0, hipMemcpyHostToDevice) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(c_delta), &h_delta, sizeof(double), 0, hipMemcpyHostToDevice) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(c_kappa), &h_kappa, sizeof(double), 0, hipMemcpyHostToDevice) );
    // coordinates
    printf("writing coordinate arrays to GPU mem.\n");
    double *R = new double[N]; // density dimension vector
    for(int j=0;j<N;j++)
        R[j] = RC-RL/2+RL*(double(j)/double(N-1));
    int bytes = 0; // size of array or vector
    // R device array
    bytes = N*sizeof(double);
    double* d_R; // R on device
    // (1) allocate
    checkCuda( hipMalloc((void**)&d_R, bytes) );
    // (2) write initial values
    checkCuda( hipMemcpy(d_R, R, bytes, hipMemcpyHostToDevice) );  
    // arrays of volumetric density and flux
    printf("writing flux and density arrays to GPU mem.\n");
    double *phi = new double[N*N];
    double *dJ = new double[N*N];
    double *J = new double[N*N];
    // write initial values (calculated from R)
    initPhi(phi,R);
    /* write initial condition to drive
    sprintf(outFileName,"initPhi.dat");
    saveArrToDrive(phi,outFileName);*/
    // device arrays
    bytes = N*N*sizeof(double);
    double *d_phi, *d_dJ, *d_J;
    // (1) allocate
    checkCuda( hipMalloc((void**)&d_phi, bytes) );
    checkCuda( hipMalloc((void**)&d_dJ, bytes) );
    checkCuda( hipMalloc((void**)&d_J, bytes) );
    // (2) write initial values
    checkCuda( hipMemcpy(d_phi, phi, bytes, hipMemcpyHostToDevice) );  
    checkCuda( hipMemset(d_dJ, 0, bytes) );
    checkCuda( hipMemset(d_J, 0, bytes) );
    printf("writing vectors to GPU mem.\n");

    /* interaction kernel */
    genConvKernel();
    sprintf(outFileName,"intKernel.dat");
    saveNVecToDrive(intKernel,outFileName,kernelN);
    bytes = kernelN*sizeof(double);
    double *d_intKernel;
    // (1) allocate
    printf("allocate intkernel.\n");
    checkCuda( hipMalloc((void**)&d_intKernel, bytes) );
    // (2) write initial values
    printf("write intkernel.\n");
    checkCuda( hipMemcpy(d_intKernel, intKernel, bytes, hipMemcpyHostToDevice) );  

    /* interaction integral */
    double* psi = new double[N]; // for gathering data from device
    double* I = new double[N]; // for gathering data from device
    bytes = N*sizeof(double);
    double *d_I;
    // (1) allocate
    printf("allocate integral.\n");
    checkCuda( hipMalloc((void**)&d_I, bytes) );
    // (2) write initial values
    printf("write integral.\n");
    checkCuda( hipMemset(d_I, 0, bytes) );

    /* psi - volume fraction */
    printf("allocate psi.\n");
    bytes = N*sizeof(double);
    double *d_psi;
    // (1) allocate
    checkCuda( hipMalloc((void**)&d_psi, bytes) );
    printf("write psi.\n");
    // (2) write initial values
    checkCuda( hipMemset(d_psi, 0, bytes) );

    /* interpolated psi */
    printf("allocate interpolated psi.\n");
    bytes = sizeof(double)*M;
    double *d_psiIntp;
    // (1) allocate
    checkCuda( hipMalloc((void**)&d_psiIntp, bytes) );
    printf("write psi.\n");
    // (2) write initial values
    checkCuda( hipMemset(d_psiIntp, 0, bytes) );

    /* interpolated I integral */
    printf("allocate interpolated I.\n");
    bytes = sizeof(double)*M;
    double *d_IIntp;
    // (1) allocate
    checkCuda( hipMalloc((void**)&d_IIntp, bytes) );
    printf("write psi.\n");
    // (2) write initial values
    checkCuda( hipMemset(d_IIntp, 0, bytes) );

    /* percoll - gradient */
    printf("allocate percoll.\n");
    double *percoll = new double[N];
    for(int k=0; k<N; k++)
        percoll[k] = 0.0;

    // percoll device array
    bytes = N*sizeof(double);
    double* d_percoll; // R on device
    // (1) allocate
    checkCuda( hipMalloc((void**)&d_percoll, bytes) );
    // (2) write initial values
    checkCuda( hipMemcpy(d_percoll, percoll, bytes, hipMemcpyHostToDevice) );
    
    // gradient wing
    printf("allocate gradient wing.\n");
    double *gradWing = new double[N];
    for(int i=0; i<N; i++)
        gradWing[i] = 0.0;
    // gradient wing device array
    bytes = N*sizeof(double);
    double* d_gradWing; // R on device
    // (1) allocate
    checkCuda( hipMalloc((void**)&d_gradWing, bytes) );
    // (2) write initial values
    checkCuda( hipMemcpy(d_gradWing, gradWing, bytes, hipMemcpyHostToDevice) );
    
    // arrays for interpolation computation
    bytes = (M-1)*sizeof(double);
    double * d_alp;
    checkCuda( hipMalloc((void**)&d_alp, bytes) );
    checkCuda( hipMemset(d_alp, 0, bytes) );
    
    // output interpolation
    double psiIntp[int(M)];
    
    printf("starting timer.\n");
    // start time measurement
    float milliseconds;
    hipEvent_t startEvent, stopEvent;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );
    printf("defining grid and starting loop.\n");
    // Kernel invocation
    int nBlocksX, nBlocksY, nThreadsX, nThreadsY;
    // grid layout, usually max threads in X dimension (1024)
 
    nThreadsX = N;
    nThreadsY = 1;
    nBlocksX = 1;
    nBlocksY = N;

    dim3 numBlocks(nBlocksX,nBlocksY);
    dim3 threadsPerBlock(nThreadsX,nThreadsY);

    dim3 numBlocksA(subDiv,1);
    dim3 threadsPerBlockA(N,1);

    dim3 numBlocksD(1,1);
    dim3 threadsPerBlockD(N,1);

    printf("N = %d, M = %d\n",N,M);
    printf("alpha = %.32e\nbeta = %.32e\n",h_alpha,h_beta);
    printf("gamma = %.32e\ndelta = %.32e\nkappa = %.32e\n",h_gamma,h_delta,h_kappa);
    printf("system size L = %.32e m\n",sysL);
    printf("increment size dz = %.32e m\n",IZ);
    printf("launching with\n nBlocksX\t| nThreadsX\t| nBlocksY\t| nThreadsY\n %d\t\t| %d\t\t| %d\t\t| %d\n",nBlocksX,nThreadsX,nBlocksY,nThreadsY);
    checkCuda( hipEventRecord(startEvent, 0) );
    // iteration loop
    int n_out = NO;
    double t = 0.0;
    for (int i = 0; i < NT; i++){
        /* integration */
        CuKernelInte <<< numBlocks, threadsPerBlock >>> (d_phi,d_psi);
        /* interpolation */
        CuKernelCmpA <<< numBlocksA, threadsPerBlockA >>> (d_psi, d_alp);
        CuKernelCmpL <<< numBlocksA, threadsPerBlockA >>> (d_psi, d_alp, d_psiIntp);
        CuKernelConv <<< numBlocksA, threadsPerBlockA >>> (d_psiIntp,d_IIntp,d_intKernel);
        CuKernelDSmp <<< numBlocksD, threadsPerBlockD >>> (d_IIntp, d_I);
        /* density gradient */
        CuKernelGrad <<< numBlocks, threadsPerBlock >>> (d_percoll, t);
        CuKernelWing <<< numBlocks, threadsPerBlock >>> (d_percoll, d_gradWing, t);
        /* iteration */
        CuKernelIter <<< numBlocks, threadsPerBlock >>> (d_phi, d_J, d_dJ, d_percoll, d_R, d_I,d_psi,d_intKernel,t,d_gradWing);
        if( (((i-1) % n_out) == 0) | (i == 1) | (i==NT)){
            // retrieve data from GPU mem
            bytes = N*N*sizeof(double);
            checkCuda( hipMemcpy(phi, d_phi, bytes, hipMemcpyDeviceToHost) );
            checkCuda( hipMemcpy(J, d_J, bytes, hipMemcpyDeviceToHost) );
            checkCuda( hipMemcpy(dJ, d_dJ, bytes, hipMemcpyDeviceToHost) );
            checkCuda( hipMemcpy(I, d_I, N*sizeof(double), hipMemcpyDeviceToHost) );
            checkCuda( hipMemcpy(psi, d_psi, N*sizeof(double), hipMemcpyDeviceToHost) );
            checkCuda( hipMemcpy(psiIntp, d_psiIntp, N*sizeof(double)*subDiv, hipMemcpyDeviceToHost) );
            checkCuda( hipMemcpy(gradWing, d_gradWing, N*sizeof(double), hipMemcpyDeviceToHost) );
            checkCuda( hipMemcpy(percoll, d_percoll, N*sizeof(double), hipMemcpyDeviceToHost) );
            //checkCuda( hipMemcpy(IIntp, d_IIntp, N*sizeof(double)*subDiv, hipMemcpyDeviceToHost) );
            // write data to file
            sprintf(outFileName,"phi_%010d.dat",i);
            saveArrToDrive(phi,outFileName);

            sprintf(outFileName,"psi_%010d.dat",i);
            saveVecToDrive(psi,outFileName);     
            
            sprintf(outFileName,"gW_%010d.dat",i);
            saveVecToDrive(gradWing,outFileName);

            sprintf(outFileName,"gP_%010d.dat",i);
            saveVecToDrive(percoll,outFileName);

            /* optional output
            sprintf(outFileName,"J_%010d.dat",i);
            saveArrToDrive(J,outFileName);
            sprintf(outFileName,"dJ_%010d.dat",i);
            saveArrToDrive(dJ,outFileName);
            sprintf(outFileName,"I_%010d.dat",i);
            saveVecToDrive(I,outFileName);
            

            sprintf(outFileName,"pit_%010d.dat",i);
            saveIntVecToDrive(psiIntp,outFileName);
            */
           
            // measure time
            checkCuda( hipEventRecord(stopEvent, 0) );
            checkCuda( hipEventSynchronize(stopEvent) );
            checkCuda( hipEventElapsedTime(&milliseconds, startEvent, stopEvent) );
            printf("step: %d/%d\n", i, NT);
            printf("runtime (sec): %.5f\n", milliseconds/1000.0);
            printf("remaining (sec): %.5f\n", milliseconds/1000.0 * (NT-i)/i);
       }
       t += IT;
    }
    printf("finished.\n\n");
    // stop timer
    checkCuda( hipEventRecord(stopEvent, 0) );
    checkCuda( hipEventSynchronize(stopEvent) );
    checkCuda( hipEventElapsedTime(&milliseconds, startEvent, stopEvent) );

    // show stats    
    printf("   total steps: %d\n", NT);
    printf("   total time (ms): %f\n", milliseconds);
    printf("   average time (ms): %f\n", milliseconds / NT);

    // delete arrays and free memory
    checkCuda( hipEventDestroy(startEvent) );
    checkCuda( hipEventDestroy(stopEvent) );

    checkCuda( hipFree(d_phi) );
    checkCuda( hipFree(d_dJ) );
    checkCuda( hipFree(d_J) );
    checkCuda( hipFree(d_R) );
    checkCuda( hipFree(d_percoll) );
    checkCuda( hipFree(d_I) );
    checkCuda( hipFree(d_intKernel) );
    checkCuda( hipFree(d_psi ) );
    checkCuda( hipFree(d_psiIntp ) );
    checkCuda( hipFree(d_IIntp ) );
    checkCuda( hipFree(d_alp ) );
    checkCuda( hipFree(d_gradWing ) );

    delete [] phi;
    delete [] dJ;
    delete [] J;
    delete [] I;
}